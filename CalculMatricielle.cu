
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>




// Initialise la matrice entre -1 et 1 
void MatrixInit (float *M, int n, int p){
 
 for (int i = 0; i< n; i++){
  for (int j = 0; j<p; j++){

   float number = rand();
   float randomValue = number / RAND_MAX;
   randomValue = 2*randomValue -1;
   
   //printf("Number %f \n",randomValue);
   
   *(M+i*p+j) = randomValue;
  }
 }
}

// Affiche la matrice sur le terminal 
void MatrixPrint(float *M, int n, int p){
 for (int i = 0; i< n; i++){
  for (int j = 0; j<p; j++){
   printf("%.2f \t", *(M+i*p+j));
   
  }
  printf("\n");
 }
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
 for (int i = 0; i< n; i++){
  for (int j = 0; j<p; j++){
   float numberM1 = *(M1 +i*p+j);
   float numberM2 = *(M2 +i*p+j);
   float numberOut = numberM1 + numberM2;
   //printf("%f \n", i*p+j);
   *(Mout +i*p+j) = numberOut;
  }
 }
}

/*
// Additionne deux matrices sur le GPU
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < p) {
        int index = i * p + j;
        Mout[index] = M1[index] + M2[index];
    }
}
*/

__global__ void cudaMatrixAdd(float *M1, float *M2, float *MoutGPU, int n, int p){
  int i = blockIdx.x;
  int j = threadIdx.x;
  

  
  float numberM1 = *(M1 +i*p+j);
  float numberM2 = *(M2 +i*p+j);
  float numberOut = numberM1 + numberM2;
  *(MoutGPU +i*p+j) = numberOut;
  
}


void MatrixMult(float *M1, float *M2, float *Mout, int n){
 for (int i = 0; i< n; i++){
  for (int j = 0; j<n; j++){
   *(Mout +i*n+j) = 0;
   for (int m = 0; m<n; m++){
    float numberM1 = *(M1 +i*n+m);
    float numberM2 = *(M2 +m*n+j);
    float numberOut = numberM1 * numberM2;
    *(Mout +i*n+j) += numberOut;
   }
  }
 }
}


__global__ void cudaMatrixMult(float *M1, float *M2, float *MoutGPU, int n){
  int i = blockIdx.x;
  int j = threadIdx.x;
  *(MoutGPU +i*n+j) = 0;
  for (int m = 0; m<n; m++){
    float numberM1 = *(M1 +i*n+m);
    float numberM2 = *(M2 +m*n+j);
    float numberOut = numberM1 * numberM2;
    *(MoutGPU +i*n+j) += numberOut;
   }
}



int main(int argc, char *argv[]) {
  int n = atoi(argv[1]);
  int p = atoi(argv[2]); 
  float  M1[n][p], M2[n][p], Mout[n][p], MoutGPU[n][p], MoutGPU2[n][p];
  
  // Avec le CPU : 
  MatrixInit(&M1[0][0], n, p);
  MatrixInit(&M2[0][0], n, p);
  MatrixInit(&Mout[0][0], n, p);
  MatrixInit(&MoutGPU[0][0], n, p);
  
  /*
  
  printf("Matrice M1 : \n");
  MatrixPrint(&M1[0][0], n, p);
  
  printf("Matrice M2 : \n");
  MatrixPrint(&M2[0][0], n, p);
  
  printf("\n");
  
  MatrixAdd(&M1[0][0], &M2[0][0], &Mout[0][0], n, p);
  printf("Matrice M1 + M2 : \n");
  MatrixPrint(&Mout[0][0], n, p);
  
  MatrixMult(&M1[0][0], &M2[0][0], &Mout[0][0], n);
  printf("\nMatrice M1 * M2 : \n");
  MatrixPrint(&Mout[0][0], n, p);
  */
  
  // Avec le GPU : 
  float *d_M1, *d_M2, *d_Mout, *d_Mout2;

 
  hipMalloc((void**)&d_M1, (n * p) * sizeof(float));
  hipMalloc((void**)&d_M2, (n * p) * sizeof(float));
  hipMalloc((void**)&d_Mout, (n * p) * sizeof(float));
  hipMalloc((void**)&d_Mout2, (n * p) * sizeof(float));

    
  hipMemcpy(d_M1, M1, (n * p) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_M2, M2, (n * p) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Mout, MoutGPU, (n * p) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Mout2, MoutGPU2, (n * p) * sizeof(float), hipMemcpyHostToDevice);
   
  cudaMatrixAdd<<<n, p>>>(d_M1, d_M2, d_Mout2, n, p);
  cudaMatrixMult<<<n, p>>>(d_M1, d_M2, d_Mout, n);
    
  hipMemcpy(MoutGPU, d_Mout, n * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(MoutGPU2, d_Mout2, n * p * sizeof(float), hipMemcpyDeviceToHost);
  
  printf("\nMatrice M1 + M2 (GPU) :\n");
  MatrixPrint(&MoutGPU2[0][0], n, p);
  
  printf("\nMatrice M1 * M2 (GPU) :\n");
  MatrixPrint(&MoutGPU[0][0], n, p);
    
  hipFree(d_M1);
  hipFree(d_M2);
  hipFree(d_Mout);
  hipFree(d_Mout2);

  exit(EXIT_SUCCESS);
}
