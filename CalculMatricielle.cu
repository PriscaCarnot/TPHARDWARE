#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>



// Initialise la matrice entre -1 et 1 
void MatrixInit (float *M, int n, int p){
 
 for (int i = 0; i< n; i++){
  for (int j = 0; j<p; j++){

   float number = rand();
   float randomValue = number / RAND_MAX;
   randomValue = 2*randomValue -1;
   
   //printf("Number %f \n",randomValue);
   
   *(M+i*p+j) = randomValue;
  }
 }
}

// Affiche la matrice sur le terminal 
void MatrixPrint(float *M, int n, int p){
 for (int i = 0; i< n; i++){
  for (int j = 0; j<p; j++){
   printf("%.2f \t", *(M+i*p+j));
   
  }
  printf("\n");
 }
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
 for (int i = 0; i< n; i++){
  for (int j = 0; j<p; j++){
   float numberM1 = *(M1 +i*p+j);
   float numberM2 = *(M2 +i*p+j);
   float numberOut = numberM1 + numberM2;
   //printf("%f \n", i*p+j);
   *(Mout +i*p+j) = numberOut;
  }
 }
}

/*
// Additionne deux matrices sur le GPU
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < p) {
        int index = i * p + j;
        Mout[index] = M1[index] + M2[index];
    }
}
*/

__global__ void cudaMatrixAdd(float *M1, float *M2, float *MoutGPU, int n, int p){
  int i = blockIdx.x;
  int j = threadIdx.x;
  

  
  float numberM1 = *(M1 +i*p+j);
  float numberM2 = *(M2 +i*p+j);
  float numberOut = numberM1 + numberM2;
  *(MoutGPU +i*p+j) = numberOut;
  
}


void MatrixMult(float *M1, float *M2, float *Mout, int n){
 for (int i = 0; i< n; i++){
  for (int j = 0; j<n; j++){
   *(Mout +i*n+j) = 0;
   for (int m = 0; m<n; m++){
    float numberM1 = *(M1 +i*n+m);
    float numberM2 = *(M2 +m*n+j);
    float numberOut = numberM1 * numberM2;
    *(Mout +i*n+j) += numberOut;
   }
  }
 }
}


__global__ void cudaMatrixMult(float *M1, float *M2, float *MoutGPU, int n){
  int i = blockIdx.x;
  int j = threadIdx.x;
  *(MoutGPU +i*n+j) = 0;
  for (int m = 0; m<n; m++){
    float numberM1 = *(M1 +i*n+m);
    float numberM2 = *(M2 +m*n+j);
    float numberOut = numberM1 * numberM2;
    *(MoutGPU +i*n+j) += numberOut;
   }
}



int main() {
  int n = 500;
  int p = 500; 
  float  M1[n][p], M2[n][p], Mout[n][p], MoutGPU[n][p], MoutGPU2[n][p];
  
  
  clock_t startCPU,endCPU;
  startCPU = clock();
    
  // Avec le CPU : 
  MatrixInit(&M1[0][0], n, p);
  MatrixInit(&M2[0][0], n, p);
  MatrixInit(&Mout[0][0], n, p);
  MatrixInit(&MoutGPU[0][0], n, p);
  
  printf("Matrice M1 : \n");
  MatrixPrint(&M1[0][0], n, p);
  
  printf("Matrice M2 : \n");
  MatrixPrint(&M2[0][0], n, p);
  
  printf("\n");
  
  MatrixAdd(&M1[0][0], &M2[0][0], &Mout[0][0], n, p);
  printf("Matrice M1 + M2 : \n");
  MatrixPrint(&Mout[0][0], n, p);
  
  MatrixMult(&M1[0][0], &M2[0][0], &Mout[0][0], n);
  printf("\nMatrice M1 * M2 : \n");
  MatrixPrint(&Mout[0][0], n, p);
  
  endCPU = clock();
  double CPUtime = ((double) (endCPU-startCPU))/CLOCKS_PER_SEC;
  printf("\n Temps d'éxécution CPU: %f sec \n",CPUtime);
  
  // Avec le GPU : 
  hipEvent_t startGPU,endGPU;
  hipEventCreate(&startGPU);
  hipEventCreate(&endGPU);
  
  
  hipEvent_t startGPU2,endGPU2;
  hipEventCreate(&startGPU2);
  hipEventCreate(&endGPU2);
  
  
  hipEvent_t startGPU3,endGPU3;
  hipEventCreate(&startGPU3);
  hipEventCreate(&endGPU3);
  
  hipEventRecord(startGPU);
  float *d_M1, *d_M2, *d_Mout, *d_Mout2;

  
  hipMalloc((void**)&d_M1, (n * p) * sizeof(float));
  hipMalloc((void**)&d_M2, (n * p) * sizeof(float));
  hipMalloc((void**)&d_Mout, (n * p) * sizeof(float));
  hipMalloc((void**)&d_Mout2, (n * p) * sizeof(float));

    
  hipMemcpy(d_M1, M1, (n * p) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_M2, M2, (n * p) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Mout, MoutGPU, (n * p) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Mout2, MoutGPU2, (n * p) * sizeof(float), hipMemcpyHostToDevice);
  
  hipEventRecord(endGPU);
  hipEventSynchronize(endGPU);
  float GPUtime;
  hipEventElapsedTime(&GPUtime, startGPU, endGPU);
  printf("\n Temps d'éxécution GPU (récup): %f msec \n",GPUtime);
  
  hipEventRecord(startGPU2); 
  cudaMatrixAdd<<<n, p>>>(d_M1, d_M2, d_Mout2, n, p);
  cudaMatrixMult<<<n,p>>>(d_M1, d_M2, d_Mout, n);
  
  hipEventRecord(endGPU2);
  hipEventSynchronize(endGPU2);
  float GPUtime2;
  hipEventElapsedTime(&GPUtime2, startGPU2, endGPU2);  
  printf("\n Temps d'éxécution GPU (exec): %f msec \n",GPUtime2);
  
  
  hipEventRecord(startGPU3); 
  hipMemcpy(MoutGPU, d_Mout, n * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(MoutGPU2, d_Mout2, n * p * sizeof(float), hipMemcpyDeviceToHost);
  
  /*
  printf("\nMatrice M1 + M2 (GPU) :\n");
  MatrixPrint(&MoutGPU2[0][0], n, p);
  
  printf("\nMatrice M1 * M2 (GPU) :\n");
  MatrixPrint(&MoutGPU[0][0], n, p);
  */
    
  hipFree(d_M1);
  hipFree(d_M2);
  hipFree(d_Mout);
  hipFree(d_Mout2);
  
  hipEventRecord(endGPU3);
  hipEventSynchronize(endGPU3);
  float GPUtime3;
  hipEventElapsedTime(&GPUtime3, startGPU3, endGPU3);
  printf("\n Temps d'éxécution GPU (envoie): %f msec \n",GPUtime3);

  exit(EXIT_SUCCESS);
}
